#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) Microsoft Corporation.
 * Licensed under the MIT License.
 *
 * MIT License
 *
 * Copyright (C) 2022-2023 Advanced Micro Devices, Inc.
 * Modifications Copyright (c) Microsoft Corporation. Licensed under the MIT License.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 *
 *******************************************************************************/

#include <algorithm>
#include <chrono>
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <functional>
#include <iostream>
#include <limits>
#include <mpi.h>
#include <string>
#include <unistd.h>
#include <vector>

#if defined(__HIP_PLATFORM_AMD__)
#include <hipblaslt/hipblaslt.h>
#include <rccl/rccl.h>
using cublasLtHalf = hipblasLtHalf;
#if defined(USE_HIPBLASLT_DATATYPE)
#define DIST_INF_HIP_DATATYPE_R_16F HIPBLASLT_R_16F
#define DIST_INF_HIP_DATATYPE_R_32F HIPBLASLT_R_32F
#elif defined(USE_HIP_DATATYPE)
#define DIST_INF_HIP_DATATYPE_R_16F HIP_R_16F
#define DIST_INF_HIP_DATATYPE_R_32F HIP_R_32F
#else
#define DIST_INF_HIP_DATATYPE_R_16F HIPBLAS_R_16F
#define DIST_INF_HIP_DATATYPE_R_32F HIPBLAS_R_32F
#endif
#if defined(USE_HIPBLAS_COMPUTETYPE)
#define DIST_INF_HIP_COMPUTETYPE_F32 HIPBLAS_COMPUTE_32F
#else
#define DIST_INF_HIP_COMPUTETYPE_F32 HIPBLASLT_COMPUTE_F32
#endif
#if HIP_VERSION >= 50700000
#include <hipblaslt/hipblaslt-ext.hpp>
#if HIP_VERSION >= 60000000
#define HIPBLASLT_GETINDEXFROMALGO(algo) hipblaslt_ext::getIndexFromAlgo(algo)
#else
static int getIndexFromAlgo(hipblasLtMatmulAlgo_t &algo) {
    int *algo_ptr = (int *)algo.data;
    if (*algo_ptr < 0) {
        return -1;
    }
    return *algo_ptr;
}
#define HIPBLASLT_GETINDEXFROMALGO(algo) getIndexFromAlgo(algo)
#endif
#endif
#else
#include <hipblaslt.h>
#include <hip/hip_fp16.h>
#include <nccl.h>
using cublasLtHalf = half;
#endif

#ifndef CHECK_CUDA_ERROR
#define CHECK_CUDA_ERROR(error)                                                                                        \
    if (error != hipSuccess) {                                                                                        \
        fprintf(stderr, "Cuda error: '%s'(%d) at %s:%d\n", hipGetErrorString(error), error, __FILE__, __LINE__);      \
        exit(-1);                                                                                                      \
    }
#endif

#ifndef CHECK_CUBLASLT_ERROR
#define CHECK_CUBLASLT_ERROR(error)                                                                                    \
    if (error != HIPBLAS_STATUS_SUCCESS) {                                                                              \
        fprintf(stderr, "cuBLASLt error(Err=%d) at %s:%d\n", error, __FILE__, __LINE__);                               \
        fprintf(stderr, "\n");                                                                                         \
        exit(-1);                                                                                                      \
    }
#endif

#ifndef CHECK_NCCL_ERROR
#define CHECK_NCCL_ERROR(error)                                                                                        \
    if (error != ncclSuccess) {                                                                                        \
        fprintf(stderr, "NCCL error(Err=%d) at %s:%d\n", error, __FILE__, __LINE__);                                   \
        fprintf(stderr, "\n");                                                                                         \
        exit(-1);                                                                                                      \
    }
#endif

static void ShowUsage(char *argv[]) {
    std::cerr
        << "Usage: " << argv[0] << " <options>\n"
        << "options:\n"
        << "\t-h, --help\t\t\t\tShow this help message\n"
        << "\t-m \t\t\tm\t\tGEMM_STRIDED argument m\n"
        << "\t-n \t\t\tn\t\tGEMM_STRIDED argument n\n"
        << "\t-k \t\t\tk \t\tGEMM_STRIDED argument k\n"
        << "\t--alpha \t\talpha \t\tGEMM_STRIDED argument alpha\n"
        << "\t--beta \t\t\tbeta \t\tGEMM_STRIDED argument beta\n"
        << "\t--num_layers \t\t\tnum_layers \t\tNumber of layers in the model\n"
        << "\t--num_warmups \t\t\tnum_warmups \t\tNumber of warmup runs\n"
        << "\t--num_iters \t\t\tnum_iters \t\tNumber of test runs\n"
        << "\t--use_cuda_graph \t\t\tuse_cuda_graph \t\tWhether to launch kernels in CUDA graph mode\n"
        << "\t--tune_gemm \t\t\ttune_gemm \t\tWhether to tune GEMM before testing. Currently only work for hipblasLt.\n"
        << std::endl;
}

static int ParseArguments(int argc, char *argv[], int64_t *m, int64_t *n, int64_t *k, float *alpha, float *beta,
                          int32_t *num_layers, int32_t *num_warmups, int32_t *num_iters, bool *use_cuda_graph,
                          bool *tune_gemm) {
    if (argc >= 2) {
        for (int i = 1; i < argc; ++i) {
            std::string arg = argv[i];

            if ((arg.at(0) == '-') || ((arg.at(0) == '-') && (arg.at(1) == '-'))) {
                if ((arg == "-h") || (arg == "--help")) {
                    return -1;
                } else if ((arg == "-m") && (i + 1 < argc)) {
                    *m = atoi(argv[++i]);
                } else if ((arg == "-n") && (i + 1 < argc)) {
                    *n = atoi(argv[++i]);
                } else if ((arg == "-k") && (i + 1 < argc)) {
                    *k = atoi(argv[++i]);
                } else if ((arg == "--alpha") && (i + 1 < argc)) {
                    *alpha = atof(argv[++i]);
                } else if ((arg == "--beta") && (i + 1 < argc)) {
                    *beta = atof(argv[++i]);
                } else if ((arg == "--num_layers") && (i + 1 < argc)) {
                    *num_layers = atoi(argv[++i]);
                } else if ((arg == "--num_warmups") && (i + 1 < argc)) {
                    *num_warmups = atoi(argv[++i]);
                } else if ((arg == "--num_iters") && (i + 1 < argc)) {
                    *num_iters = atoi(argv[++i]);
                } else if (arg == "--use_cuda_graph") {
#if (NCCL_MAJOR > 2 || (NCCL_MAJOR >= 2 && NCCL_MINOR >= 9)) && (CUDART_VERSION >= 11030 || HIP_VERSION >= 50221310)
                    *use_cuda_graph = true;
#else
                    *use_cuda_graph = false;
                    std::cerr << "error with " << arg << std::endl;
                    std::cerr << "not supported by current environment" << std::endl << std::endl;
                    return -1;
#endif
                } else if (arg == "--tune_gemm") {
                    *tune_gemm = true;
                } else {
                    std::cerr << "error with " << arg << std::endl;
                    std::cerr << "do not recognize option" << std::endl << std::endl;
                    return -1;
                }
            } else {
                std::cerr << "error with " << arg << std::endl;
                std::cerr << "option must start with - or --" << std::endl << std::endl;
                return -1;
            }
        }
    }
    return 0;
}

void InitializeABCDEF(std::vector<cublasLtHalf> &ha, int64_t size_a, std::vector<cublasLtHalf> &hb, int64_t size_b,
                      std::vector<cublasLtHalf> &hc, int64_t size_c, std::vector<cublasLtHalf> &hd, int64_t size_d,
                      std::vector<cublasLtHalf> &he, int64_t size_e, std::vector<cublasLtHalf> &hf, int64_t size_f) {
    srand(1);
    for (int i = 0; i < size_a; ++i) {
        ha[i] = static_cast<cublasLtHalf>((rand() % 7) - 3);
    }
    for (int i = 0; i < size_b; ++i) {
        hb[i] = static_cast<cublasLtHalf>((rand() % 7) - 3);
    }
    for (int i = 0; i < size_c; ++i) {
        hc[i] = static_cast<cublasLtHalf>((rand() % 7) - 3);
    }
    for (int i = 0; i < size_d; ++i) {
        hd[i] = static_cast<cublasLtHalf>((rand() % 7) - 3);
    }
    for (int i = 0; i < size_e; ++i) {
        he[i] = static_cast<cublasLtHalf>((rand() % 7) - 3);
    }
    for (int i = 0; i < size_f; ++i) {
        hf[i] = static_cast<cublasLtHalf>((rand() % 7) - 3);
    }
}

#if defined(__HIP_PLATFORM_AMD__) && HIP_VERSION >= 50700000
// Tune GEMM algorithm in local rank.
// Write <0 to ret_algo_time_in_ms if nothing found.
// Write >=0 to ret_algo_time_in_ms and write ret_algo if something is found.
void TuneHipblasLtGemmLocal(const hipblasLtHandle_t &handle, const hipblasLtMatmulDesc_t &matmul, float alpha, void *da,
                            const hipblasLtMatrixLayout_t &matA, void *db, const hipblasLtMatrixLayout_t &matB,
                            float beta, void *dc, const hipblasLtMatrixLayout_t &matC, void *dd,
                            const hipblasLtMatrixLayout_t &matD, void *d_workspace, uint64_t workspace_size,
                            const hipStream_t &stream, int rank, int num_ranks, hipblasLtMatmulAlgo_t *ret_algo,
                            float *ret_algo_time_in_ms) {
    std::vector<hipblasLtMatmulHeuristicResult_t> gemm_heuristics;
    // Get all possible algorithms
    CHECK_CUBLASLT_ERROR(hipblaslt_ext::getAllAlgos(
        handle, hipblaslt_ext::GemmType::HIPBLASLT_GEMM, HIPBLAS_OP_N, HIPBLAS_OP_N, DIST_INF_HIP_DATATYPE_R_16F,
        DIST_INF_HIP_DATATYPE_R_16F, DIST_INF_HIP_DATATYPE_R_16F, DIST_INF_HIP_DATATYPE_R_16F,
        DIST_INF_HIP_COMPUTETYPE_F32, gemm_heuristics));
    // Make sure the algorithm order is deterministic
    std::sort(gemm_heuristics.begin(), gemm_heuristics.end(),
              [](hipblasLtMatmulHeuristicResult_t &a, hipblasLtMatmulHeuristicResult_t &b) {
                  return HIPBLASLT_GETINDEXFROMALGO(a.algo) < HIPBLASLT_GETINDEXFROMALGO(b.algo);
              });
    // Timing utilities
    hipEvent_t start_event;
    hipEvent_t end_event;
    const int kNumWarmups = 10;
    const int kNumTestRuns = 100;
    *ret_algo_time_in_ms = -1;
    // Benchmark all algorithms in given shape
    CHECK_CUDA_ERROR(hipEventCreate(&start_event));
    CHECK_CUDA_ERROR(hipEventCreate(&end_event));
    // Partition work evenly into different ranks
    for (size_t algo_idx = rank; algo_idx < gemm_heuristics.size(); algo_idx += num_ranks) {
        auto &algo = gemm_heuristics[algo_idx].algo;
        size_t ret_workspace_size = 0;
        auto status = hipblaslt_ext::matmulIsAlgoSupported(handle, matmul, &alpha, matA, matB, &beta, matC, matD, algo,
                                                           ret_workspace_size);
        if (status != HIPBLAS_STATUS_SUCCESS || ret_workspace_size >= workspace_size) {
            continue;
        }
        for (int i = 0; i < kNumWarmups; i++) {
            CHECK_CUBLASLT_ERROR(hipblasLtMatmul(handle, matmul, &alpha, da, matA, db, matB, &beta, dc, matC, dd, matD,
                                                 &algo, d_workspace, workspace_size, stream));
        }
        CHECK_CUDA_ERROR(hipEventRecord(start_event, stream));
        for (int i = 0; i < kNumTestRuns; i++) {
            CHECK_CUBLASLT_ERROR(hipblasLtMatmul(handle, matmul, &alpha, da, matA, db, matB, &beta, dc, matC, dd, matD,
                                                 &algo, d_workspace, workspace_size, stream));
        }
        CHECK_CUDA_ERROR(hipEventRecord(end_event, stream));
        CHECK_CUDA_ERROR(hipStreamSynchronize(stream));
        float time_in_ms = 0;
        CHECK_CUDA_ERROR(hipEventElapsedTime(&time_in_ms, start_event, end_event));
        time_in_ms /= kNumTestRuns;
        if (*ret_algo_time_in_ms < 0 || time_in_ms < *ret_algo_time_in_ms) {
            *ret_algo = algo;
            *ret_algo_time_in_ms = time_in_ms;
        }
    }
    CHECK_CUDA_ERROR(hipEventDestroy(start_event));
    CHECK_CUDA_ERROR(hipEventDestroy(end_event));
}

// Select global best GEMM algorithms across ranks. Write global_algo if something is found.
void TuneHipblasLtGemmGlobal(int num_ranks, const hipblasLtMatmulAlgo_t &local_algo, float local_time_in_ms,
                             hipblasLtMatmulAlgo_t *global_algo) {
    std::vector<hipblasLtMatmulAlgo_t> coll_algo(num_ranks);
    std::vector<float> coll_time_in_ms(num_ranks);
    MPI_Allgather(&local_algo, sizeof(local_algo), MPI_BYTE, coll_algo.data(), sizeof(local_algo), MPI_BYTE,
                  MPI_COMM_WORLD);
    MPI_Allgather(&local_time_in_ms, sizeof(local_time_in_ms), MPI_BYTE, coll_time_in_ms.data(),
                  sizeof(local_time_in_ms), MPI_BYTE, MPI_COMM_WORLD);
    float min_time_in_ms = -1;
    for (int i = 0; i < num_ranks; i++) {
        if (coll_time_in_ms[i] >= 0 && (min_time_in_ms < 0 || coll_time_in_ms[i] < min_time_in_ms)) {
            min_time_in_ms = coll_time_in_ms[i];
            *global_algo = coll_algo[i];
        }
    }
}
#endif

// B[m, k] * A[k, n] + C[m, n] = D[m, n]
// E[k, m] * D[m, n] + F[k, n] = G[k, n]
void TestModel(int64_t m, int64_t n, int64_t k, float alpha, float beta, int32_t num_layers, int32_t num_warmups,
               int32_t num_iters, bool use_cuda_graph, bool tune_gemm, ncclComm_t nccl_comm, int rank, int num_ranks) {
    const int kNcclBufAlignment = 512;

    int size_a = k * n;
    int size_b = m * k;
    int size_c = m * n;
    int size_d = m * n;
    int size_e = k * m;
    int size_f = k * n;
    int size_g = (k * n + kNcclBufAlignment - 1) / kNcclBufAlignment * kNcclBufAlignment;

    // Naming: da is in GPU (device) memory. ha is in CPU (host) memory
    std::vector<cublasLtHalf> ha(size_a);
    std::vector<cublasLtHalf> hb(size_b);
    std::vector<cublasLtHalf> hc(size_c);
    std::vector<cublasLtHalf> hd(size_d);
    std::vector<cublasLtHalf> he(size_e);
    std::vector<cublasLtHalf> hf(size_f);
    std::vector<cublasLtHalf> hg(size_g);

    // initial data on host
    InitializeABCDEF(ha, size_a, hb, size_b, hc, size_c, hd, size_d, he, size_e, hf, size_f);

    // allocate memory on device
    void *da, *db, *dc, *dd, *de, *df, *dg;

    // Create stream
    hipStream_t stream = nullptr;
    CHECK_CUDA_ERROR(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

    CHECK_CUDA_ERROR(hipMalloc(&da, size_a * sizeof(cublasLtHalf)));
    CHECK_CUDA_ERROR(hipMalloc(&db, size_b * sizeof(cublasLtHalf)));
    CHECK_CUDA_ERROR(hipMalloc(&dc, size_c * sizeof(cublasLtHalf)));
    CHECK_CUDA_ERROR(hipMalloc(&dd, size_d * sizeof(cublasLtHalf)));
    CHECK_CUDA_ERROR(hipMalloc(&de, size_e * sizeof(cublasLtHalf)));
    CHECK_CUDA_ERROR(hipMalloc(&df, size_f * sizeof(cublasLtHalf)));
    CHECK_CUDA_ERROR(hipMalloc(&dg, size_g * sizeof(cublasLtHalf)));
    // copy matrices from host to device
    CHECK_CUDA_ERROR(hipMemcpy(da, ha.data(), sizeof(cublasLtHalf) * size_a, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(db, hb.data(), sizeof(cublasLtHalf) * size_b, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(dc, hc.data(), sizeof(cublasLtHalf) * size_c, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(dd, hd.data(), sizeof(cublasLtHalf) * size_d, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(de, he.data(), sizeof(cublasLtHalf) * size_e, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(df, hf.data(), sizeof(cublasLtHalf) * size_f, hipMemcpyHostToDevice));

#if defined(__HIP_PLATFORM_AMD__)
    uint64_t workspace_size = 256 * 1024 * 1024; // max workspace size allowed for hipblaslt
#else
    uint64_t workspace_size = 1024 * 1024;
#endif
    void *d_workspace;
    CHECK_CUDA_ERROR(hipMalloc(&d_workspace, workspace_size));
    int returnedAlgoCount = 0;

    // cublasLt is not well supported by ROCm hipify tools, explicitly define ROCm logic instead.
#if defined(__HIP_PLATFORM_AMD__)
    hipblasLtHandle_t handle;
    hipblasLtMatrixLayout_t matA, matB, matC, matD, matE, matF, matG;
    hipblasLtMatmulDesc_t matmul1, matmul2;
    hipblasLtMatmulPreference_t pref;

    CHECK_CUBLASLT_ERROR(hipblasLtCreate(&handle));

    CHECK_CUBLASLT_ERROR(hipblasLtMatrixLayoutCreate(&matA, DIST_INF_HIP_DATATYPE_R_16F, k, n, k));
    CHECK_CUBLASLT_ERROR(hipblasLtMatrixLayoutCreate(&matB, DIST_INF_HIP_DATATYPE_R_16F, m, k, m));
    CHECK_CUBLASLT_ERROR(hipblasLtMatrixLayoutCreate(&matC, DIST_INF_HIP_DATATYPE_R_16F, m, n, m));
    CHECK_CUBLASLT_ERROR(hipblasLtMatrixLayoutCreate(&matD, DIST_INF_HIP_DATATYPE_R_16F, m, n, m));
    CHECK_CUBLASLT_ERROR(hipblasLtMatrixLayoutCreate(&matE, DIST_INF_HIP_DATATYPE_R_16F, k, m, k));
    CHECK_CUBLASLT_ERROR(hipblasLtMatrixLayoutCreate(&matF, DIST_INF_HIP_DATATYPE_R_16F, k, n, k));
    CHECK_CUBLASLT_ERROR(hipblasLtMatrixLayoutCreate(&matG, DIST_INF_HIP_DATATYPE_R_16F, k, n, k));

    CHECK_CUBLASLT_ERROR(
        hipblasLtMatmulDescCreate(&matmul1, DIST_INF_HIP_COMPUTETYPE_F32, DIST_INF_HIP_DATATYPE_R_32F));
    CHECK_CUBLASLT_ERROR(
        hipblasLtMatmulDescCreate(&matmul2, DIST_INF_HIP_COMPUTETYPE_F32, DIST_INF_HIP_DATATYPE_R_32F));

    hipblasOperation_t trans = HIPBLAS_OP_N;
    CHECK_CUBLASLT_ERROR(
        hipblasLtMatmulDescSetAttribute(matmul1, HIPBLASLT_MATMUL_DESC_TRANSA, &trans, sizeof(int32_t)));
    CHECK_CUBLASLT_ERROR(
        hipblasLtMatmulDescSetAttribute(matmul1, HIPBLASLT_MATMUL_DESC_TRANSB, &trans, sizeof(int32_t)));
    CHECK_CUBLASLT_ERROR(
        hipblasLtMatmulDescSetAttribute(matmul2, HIPBLASLT_MATMUL_DESC_TRANSA, &trans, sizeof(int32_t)));
    CHECK_CUBLASLT_ERROR(
        hipblasLtMatmulDescSetAttribute(matmul2, HIPBLASLT_MATMUL_DESC_TRANSB, &trans, sizeof(int32_t)));

    // Set User Preference attributes
    CHECK_CUBLASLT_ERROR(hipblasLtMatmulPreferenceCreate(&pref));
    CHECK_CUBLASLT_ERROR(hipblasLtMatmulPreferenceSetAttribute(pref, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES,
                                                               &workspace_size, sizeof(workspace_size)));

    // Get Heuristic results
    hipblasLtMatmulHeuristicResult_t heuristicResult1[1] = {0};
    hipblasLtMatmulHeuristicResult_t heuristicResult2[1] = {0};
    // B[m, k] * A[k, n] + C[m, n] = D[m, n]
    // E[k, m] * D[m, n] + F[k, n] = G[k, n]
    CHECK_CUBLASLT_ERROR(hipblasLtMatmulAlgoGetHeuristic(handle, matmul1, matB, matA, matC, matD, pref, 1,
                                                         heuristicResult1, &returnedAlgoCount));
    hipblasLtMatmulAlgo_t algo1 = heuristicResult1[0].algo;
    CHECK_CUBLASLT_ERROR(hipblasLtMatmulAlgoGetHeuristic(handle, matmul2, matE, matD, matF, matG, pref, 1,
                                                         heuristicResult2, &returnedAlgoCount));
    hipblasLtMatmulAlgo_t algo2 = heuristicResult2[0].algo;
#if HIP_VERSION >= 50700000
    if (tune_gemm) {
        hipblasLtMatmulAlgo_t ret_algo;
        float ret_algo_time_in_ms;
        TuneHipblasLtGemmLocal(handle, matmul1, alpha, db, matB, da, matA, beta, dc, matC, dd, matD, d_workspace,
                               workspace_size, stream, rank, num_ranks, &ret_algo, &ret_algo_time_in_ms);
        TuneHipblasLtGemmGlobal(num_ranks, ret_algo, ret_algo_time_in_ms, &algo1);
        TuneHipblasLtGemmLocal(handle, matmul2, alpha, de, matE, dd, matD, beta, df, matF, dg, matG, d_workspace,
                               workspace_size, stream, rank, num_ranks, &ret_algo, &ret_algo_time_in_ms);
        TuneHipblasLtGemmGlobal(num_ranks, ret_algo, ret_algo_time_in_ms, &algo2);
    }
#endif
#else
    hipblasLtHandle_t handle;
    hipblasLtMatrixLayout_t matA, matB, matC, matD, matE, matF, matG;
    hipblasLtMatmulDesc_t matmul1, matmul2;
    hipblasLtMatmulPreference_t pref;
    CHECK_CUBLASLT_ERROR(hipblasLtCreate(&handle));

    CHECK_CUBLASLT_ERROR(hipblasLtMatrixLayoutCreate(&matA, HIP_R_16F, k, n, k));
    CHECK_CUBLASLT_ERROR(hipblasLtMatrixLayoutCreate(&matB, HIP_R_16F, m, k, m));
    CHECK_CUBLASLT_ERROR(hipblasLtMatrixLayoutCreate(&matC, HIP_R_16F, m, n, m));
    CHECK_CUBLASLT_ERROR(hipblasLtMatrixLayoutCreate(&matD, HIP_R_16F, m, n, m));
    CHECK_CUBLASLT_ERROR(hipblasLtMatrixLayoutCreate(&matE, HIP_R_16F, k, m, k));
    CHECK_CUBLASLT_ERROR(hipblasLtMatrixLayoutCreate(&matF, HIP_R_16F, k, n, k));
    CHECK_CUBLASLT_ERROR(hipblasLtMatrixLayoutCreate(&matG, HIP_R_16F, k, n, k));

    CHECK_CUBLASLT_ERROR(hipblasLtMatmulDescCreate(&matmul1, HIPBLAS_COMPUTE_16F, HIP_R_32F));
    CHECK_CUBLASLT_ERROR(hipblasLtMatmulDescCreate(&matmul2, HIPBLAS_COMPUTE_16F, HIP_R_32F));

    hipblasOperation_t trans = HIPBLAS_OP_N;
    CHECK_CUBLASLT_ERROR(hipblasLtMatmulDescSetAttribute(matmul1, HIPBLASLT_MATMUL_DESC_TRANSA, &trans, sizeof(int32_t)));
    CHECK_CUBLASLT_ERROR(hipblasLtMatmulDescSetAttribute(matmul1, HIPBLASLT_MATMUL_DESC_TRANSB, &trans, sizeof(int32_t)));
    CHECK_CUBLASLT_ERROR(hipblasLtMatmulDescSetAttribute(matmul2, HIPBLASLT_MATMUL_DESC_TRANSA, &trans, sizeof(int32_t)));
    CHECK_CUBLASLT_ERROR(hipblasLtMatmulDescSetAttribute(matmul2, HIPBLASLT_MATMUL_DESC_TRANSB, &trans, sizeof(int32_t)));

    // Set User Preference attributes
    CHECK_CUBLASLT_ERROR(hipblasLtMatmulPreferenceCreate(&pref));
    CHECK_CUBLASLT_ERROR(hipblasLtMatmulPreferenceSetAttribute(pref, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES,
                                                              &workspace_size, sizeof(workspace_size)));

    // Get Heuristic results
    hipblasLtMatmulHeuristicResult_t heuristicResult1[1] = {0};
    hipblasLtMatmulHeuristicResult_t heuristicResult2[1] = {0};
    // B[m, k] * A[k, n] + C[m, n] = D[m, n]
    // E[k, m] * D[m, n] + F[k, n] = G[k, n]
    CHECK_CUBLASLT_ERROR(hipblasLtMatmulAlgoGetHeuristic(handle, matmul1, matB, matA, matC, matD, pref, 1,
                                                        heuristicResult1, &returnedAlgoCount));
    CHECK_CUBLASLT_ERROR(hipblasLtMatmulAlgoGetHeuristic(handle, matmul2, matE, matD, matF, matG, pref, 1,
                                                        heuristicResult2, &returnedAlgoCount));
#endif

    auto model_forward = [&] {
        for (int j = 0; j < num_layers; j++) {
            // B[m, k] * A[k, n] + C[m, n] = D[m, n]
            // E[k, m] * D[m, n] + F[k, n] = G[k, n]
            // cublasLt is not well supported by ROCm hipify tools, explicitly define ROCm logic instead.
#if defined(__HIP_PLATFORM_AMD__)
            CHECK_CUBLASLT_ERROR(hipblasLtMatmul(handle, matmul1, &alpha, db, matB, da, matA, &beta, dc, matC, dd, matD,
                                                 &algo1, d_workspace, workspace_size, stream));
            CHECK_CUBLASLT_ERROR(hipblasLtMatmul(handle, matmul2, &alpha, de, matE, dd, matD, &beta, df, matF, dg, matG,
                                                 &algo2, d_workspace, workspace_size, stream));
#else
            CHECK_CUBLASLT_ERROR(hipblasLtMatmul(handle, matmul1, &alpha, db, matB, da, matA, &beta, dc, matC, dd, matD,
                                                &heuristicResult1[0].algo, d_workspace, workspace_size, stream));
            CHECK_CUBLASLT_ERROR(hipblasLtMatmul(handle, matmul2, &alpha, de, matE, dd, matD, &beta, df, matF, dg, matG,
                                                &heuristicResult2[0].algo, d_workspace, workspace_size, stream));
#endif
            CHECK_NCCL_ERROR(ncclAllReduce(dg, dg, size_g, ncclFloat16, ncclSum, nccl_comm, stream));
        }
    };

#if (NCCL_MAJOR > 2 || (NCCL_MAJOR >= 2 && NCCL_MINOR >= 9)) && (CUDART_VERSION >= 11030 || HIP_VERSION >= 50221310)
    hipGraph_t graph;
    hipGraphExec_t instance;
    if (use_cuda_graph) {
        CHECK_CUDA_ERROR(hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal));
        model_forward();
        CHECK_CUDA_ERROR(hipStreamEndCapture(stream, &graph));
        CHECK_CUDA_ERROR(hipGraphInstantiate(&instance, graph, NULL, NULL, 0));
    }
#endif

    std::chrono::steady_clock::time_point start_time, stop_time;
    std::vector<double> step_times(num_iters, 0.);
    for (int i = 0; i < num_warmups + num_iters; ++i) {
        if (i >= num_warmups) {
            start_time = std::chrono::steady_clock::now();
        }
#if (NCCL_MAJOR > 2 || (NCCL_MAJOR >= 2 && NCCL_MINOR >= 9)) && (CUDART_VERSION >= 11030 || HIP_VERSION >= 50221310)
        if (use_cuda_graph) {
            CHECK_CUDA_ERROR(hipGraphLaunch(instance, stream));
        } else {
            model_forward();
        }
#else
        model_forward();
#endif
        CHECK_CUDA_ERROR(hipStreamSynchronize(stream));
        if (i >= num_warmups) {
            stop_time = std::chrono::steady_clock::now();
            double step_time = std::chrono::duration_cast<std::chrono::nanoseconds>(stop_time - start_time).count();
            step_times[i - num_warmups] = step_time;
        }
    }
    for (int i = 0; i < num_iters; i++) {
        fprintf(stdout, "Latency of step %d: %g ms\n", i, step_times[i] / 1e6);
    }

#if (NCCL_MAJOR > 2 || (NCCL_MAJOR >= 2 && NCCL_MINOR >= 9)) && (CUDART_VERSION >= 11030 || HIP_VERSION >= 50221310)
    // Destroy graph
    if (use_cuda_graph) {
        CHECK_CUDA_ERROR(hipGraphExecDestroy(instance));
        CHECK_CUDA_ERROR(hipGraphDestroy(graph));
    }
#endif

    // Destroy stream
    CHECK_CUDA_ERROR(hipStreamDestroy(stream));

    CHECK_CUDA_ERROR(hipFree(da));
    CHECK_CUDA_ERROR(hipFree(db));
    CHECK_CUDA_ERROR(hipFree(dc));
    CHECK_CUDA_ERROR(hipFree(dd));
    CHECK_CUDA_ERROR(hipFree(de));
    CHECK_CUDA_ERROR(hipFree(df));
    CHECK_CUDA_ERROR(hipFree(dg));
    CHECK_CUDA_ERROR(hipFree(d_workspace));
    // cublasLt is not well supported by ROCm hipify tools, explicitly define ROCm logic instead.
#if defined(__HIP_PLATFORM_AMD__)
    CHECK_CUBLASLT_ERROR(hipblasLtMatmulPreferenceDestroy(pref));
    CHECK_CUBLASLT_ERROR(hipblasLtMatmulDescDestroy(matmul1));
    CHECK_CUBLASLT_ERROR(hipblasLtMatmulDescDestroy(matmul2));
    CHECK_CUBLASLT_ERROR(hipblasLtMatrixLayoutDestroy(matA));
    CHECK_CUBLASLT_ERROR(hipblasLtMatrixLayoutDestroy(matB));
    CHECK_CUBLASLT_ERROR(hipblasLtMatrixLayoutDestroy(matC));
    CHECK_CUBLASLT_ERROR(hipblasLtMatrixLayoutDestroy(matD));
    CHECK_CUBLASLT_ERROR(hipblasLtMatrixLayoutDestroy(matE));
    CHECK_CUBLASLT_ERROR(hipblasLtMatrixLayoutDestroy(matF));
    CHECK_CUBLASLT_ERROR(hipblasLtMatrixLayoutDestroy(matG));
    CHECK_CUBLASLT_ERROR(hipblasLtDestroy(handle));
#else
    CHECK_CUBLASLT_ERROR(hipblasLtMatmulPreferenceDestroy(pref));
    CHECK_CUBLASLT_ERROR(hipblasLtMatmulDescDestroy(matmul1));
    CHECK_CUBLASLT_ERROR(hipblasLtMatmulDescDestroy(matmul2));
    CHECK_CUBLASLT_ERROR(hipblasLtMatrixLayoutDestroy(matA));
    CHECK_CUBLASLT_ERROR(hipblasLtMatrixLayoutDestroy(matB));
    CHECK_CUBLASLT_ERROR(hipblasLtMatrixLayoutDestroy(matC));
    CHECK_CUBLASLT_ERROR(hipblasLtMatrixLayoutDestroy(matD));
    CHECK_CUBLASLT_ERROR(hipblasLtMatrixLayoutDestroy(matE));
    CHECK_CUBLASLT_ERROR(hipblasLtMatrixLayoutDestroy(matF));
    CHECK_CUBLASLT_ERROR(hipblasLtMatrixLayoutDestroy(matG));
    CHECK_CUBLASLT_ERROR(hipblasLtDestroy(handle));
#endif

    return;
}

int main(int argc, char *argv[]) {
    // Init MPI
    int comm_rank, comm_size;
    MPI_Init(NULL, NULL);
    MPI_Comm_rank(MPI_COMM_WORLD, &comm_rank);
    MPI_Comm_size(MPI_COMM_WORLD, &comm_size);

    // Init NCCL
    int num_local_ranks = 0;
    ncclComm_t nccl_comm;
    ncclUniqueId nccl_id;
    if (comm_rank == 0) {
        CHECK_NCCL_ERROR(ncclGetUniqueId(&nccl_id));
    }
    MPI_Bcast(&nccl_id, sizeof(ncclUniqueId), MPI_BYTE, 0, MPI_COMM_WORLD);
    CHECK_CUDA_ERROR(hipGetDeviceCount(&num_local_ranks))
    CHECK_CUDA_ERROR(hipSetDevice(comm_rank % num_local_ranks));
    CHECK_NCCL_ERROR(ncclCommInitRank(&nccl_comm, comm_size, nccl_id, comm_rank));

    // Init parameters with default values
    int64_t m = 80;
    int64_t n = 128;
    int64_t k = 128;
    float alpha = 1;
    float beta = 1;
    int32_t num_layers = 50;
    int32_t num_warmups = 20;
    int32_t num_iters = 100;
    bool use_cuda_graph = false;
    bool tune_gemm = false;

    if (ParseArguments(argc, argv, &m, &n, &k, &alpha, &beta, &num_layers, &num_warmups, &num_iters, &use_cuda_graph,
                       &tune_gemm)) {
        ShowUsage(argv);
        return -1;
    }

    fprintf(stdout,
            "Parameters: m=%ld, n=%ld, k=%ld, alpha=%f, beta=%f, num_layers=%d, num_warmups=%d, num_iters=%d, "
            "use_cuda_graph=%d, tune_gemm=%d\n",
            m, n, k, alpha, beta, num_layers, num_warmups, num_iters, (int)use_cuda_graph, (int)tune_gemm);

    TestModel(m, n, k, alpha, beta, num_layers, num_warmups, num_iters, use_cuda_graph, tune_gemm, nccl_comm, comm_rank,
              comm_size);

    CHECK_NCCL_ERROR(ncclCommDestroy(nccl_comm));

    MPI_Finalize();

    return 0;
}
