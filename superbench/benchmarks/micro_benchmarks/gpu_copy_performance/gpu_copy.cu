// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

// GPU copy benchmark tests dtoh/htod/dtod data transfer bandwidth by GPU SM/DMA.

#include <cstdio>
#include <cstring>
#include <string>
#include <vector>

#include <getopt.h>
#include <numa.h>

#include <hip/hip_runtime.h>


// Arguments for each sub benchmark run.
struct SubBenchArgs {
    // Whether source device is GPU.
    bool is_src_dev_gpu = false;

    // Whether destination device is GPU.
    bool is_dst_dev_gpu = false;

    // GPU IDs for source device (if applicable).
    int src_gpu_id = 0;

    // GPU IDs for destination device (if applicable).
    int dst_gpu_id = 0;

    // GPU IDs for worker device.
    int worker_gpu_id = 0;

    // Original data buffer.
    uint8_t *data_buf = nullptr;

    // Buffer to validate the correctness of data transfer.
    uint8_t *check_buf = nullptr;

    // Host pointer of the data buffer on source device.
    uint8_t *src_dev_host_buf_ptr = nullptr;

    // GPU pointer of the data buffer on source devices.
    uint8_t *src_dev_gpu_buf_ptr = nullptr;

    // Host pointer of the data buffer on destination device.
    uint8_t *dst_dev_host_buf_ptr = nullptr;

    // GPU pointer of the data buffer on destination devices.
    uint8_t *dst_dev_gpu_buf_ptr = nullptr;

    // CUDA stream to be used.
    hipStream_t stream;

    // CUDA event to record start time.
    hipEvent_t start_event;

    // CUDA event to record end time.
    hipEvent_t end_event;
};

// Arguments for each benchmark run.
struct BenchArgs {
    // Max number of sub benchmarks.
    static const int kMaxNumSubs = 2;

    // Number of sub benchmarks in this benchmark run.
    // 1 for unidirectional, 2 for bidirectional.
    int num_subs = 0;

    // NUMA node under which the benchmark is done.
    uint64_t numa_id = 0;

    // Data buffer size used.
    uint64_t size = 0;

    // Number of warm up rounds to run.
    uint64_t num_warm_up = 0;

    // Number of loops to run.
    uint64_t num_loops = 0;

    // Uses SM copy, otherwise DMA copy.
    bool is_sm_copy = false;

    // Whether check data after copy.
    bool check_data = false;

    // Sub-benchmarks in parallel.
    SubBenchArgs subs[kMaxNumSubs];
};

// Options accepted by this program.
struct Opts {
    // Data buffer size for copy benchmark.
    uint64_t size = 0;

    // Number of warm up rounds to run.
    uint64_t num_warm_up = 0;

    // Number of loops to run.
    uint64_t num_loops = 0;

    // Whether GPU SM copy needs to be evaluated.
    bool sm_copy_enabled = false;

    // Whether GPU DMA copy needs to be evaluated.
    bool dma_copy_enabled = false;

    // Whether host-to-device transfer needs to be evaluated.
    bool htod_enabled = false;

    // Whether device-to-host transfer needs to be evaluated.
    bool dtoh_enabled = false;

    // Whether device-to-device transfer needs to be evaluated.
    bool dtod_enabled = false;

    // Whether one-to-all (device) transfer needs to be evaluated.
    bool one_to_all_enabled = false;

    // Whether all-to-one (device) transfer needs to be evaluated.
    bool all_to_one_enabled = false;

    // Whether all-to-all (device) transfer needs to be evaluated.
    bool all_to_all_enabled = false;

    // Whether bidirectional transfer is enabled.
    bool bidirectional_enabled = false;

    // Number of thread blocks per rank in one-to-all/all-to-one/all-to-all tests.
    uint64_t all_to_all_num_thread_blocks_per_rank = 8;

    // Thread block size in one-to-all/all-to-one/all-to-all tests.
    uint64_t all_to_all_thread_block_size = 512;

    // Whether check data after copy.
    bool check_data = false;
};

// Print usage of this program.
void PrintUsage() {
    printf("Usage: gpu_copy "
           "--size <size> "
           "--num_warm_up <num_warm_up> "
           "--num_loops <num_loops> "
           "[--all_to_all_num_thread_blocks_per_rank <all_to_all_num_thread_blocks_per_rank>] "
           "[--all_to_all_thread_block_size <all_to_all_thread_block_size>] "
           "[--sm_copy] "
           "[--dma_copy] "
           "[--htod] "
           "[--dtoh] "
           "[--dtod] "
           "[--one_to_all] "
           "[--all_to_one] "
           "[--all_to_all] "
           "[--bidirectional] "
           "[--check_data]\n");
}

// Parse options of this program.
int ParseOpts(int argc, char **argv, Opts *opts) {
    enum class OptIdx {
        kSize,
        kNumWarmUp,
        kNumLoops,
        kAllToAllNumThreadBlocksPerRank,
        kAllToAllThreadBlockSize,
        kEnableSmCopy,
        kEnableDmaCopy,
        kEnableHToD,
        kEnableDToH,
        kEnableDToD,
        kEnableOneToAll,
        kEnableAllToOne,
        kEnableAllToAll,
        kEnableBidirectional,
        kEnableCheckData
    };
    const struct option options[] = {
        {"size", required_argument, nullptr, static_cast<int>(OptIdx::kSize)},
        {"num_warm_up", required_argument, nullptr, static_cast<int>(OptIdx::kNumWarmUp)},
        {"num_loops", required_argument, nullptr, static_cast<int>(OptIdx::kNumLoops)},
        {"all_to_all_num_thread_blocks_per_rank", required_argument, nullptr,
         static_cast<int>(OptIdx::kAllToAllNumThreadBlocksPerRank)},
        {"all_to_all_thread_block_size", required_argument, nullptr,
         static_cast<int>(OptIdx::kAllToAllThreadBlockSize)},
        {"sm_copy", no_argument, nullptr, static_cast<int>(OptIdx::kEnableSmCopy)},
        {"dma_copy", no_argument, nullptr, static_cast<int>(OptIdx::kEnableDmaCopy)},
        {"htod", no_argument, nullptr, static_cast<int>(OptIdx::kEnableHToD)},
        {"dtoh", no_argument, nullptr, static_cast<int>(OptIdx::kEnableDToH)},
        {"dtod", no_argument, nullptr, static_cast<int>(OptIdx::kEnableDToD)},
        {"one_to_all", no_argument, nullptr, static_cast<int>(OptIdx::kEnableOneToAll)},
        {"all_to_one", no_argument, nullptr, static_cast<int>(OptIdx::kEnableAllToOne)},
        {"all_to_all", no_argument, nullptr, static_cast<int>(OptIdx::kEnableAllToAll)},
        {"bidirectional", no_argument, nullptr, static_cast<int>(OptIdx::kEnableBidirectional)},
        {"check_data", no_argument, nullptr, static_cast<int>(OptIdx::kEnableCheckData)}};
    int getopt_ret = 0;
    int opt_idx = 0;
    bool size_specified = false;
    bool num_warm_up_specified = false;
    bool num_loops_specified = false;
    bool parse_err = false;
    while (true) {
        getopt_ret = getopt_long(argc, argv, "", options, &opt_idx);
        if (getopt_ret == -1) {
            if (!size_specified || !num_warm_up_specified || !num_loops_specified) {
                parse_err = true;
            }
            break;
        } else if (getopt_ret == '?') {
            parse_err = true;
            break;
        }
        switch (opt_idx) {
        case static_cast<int>(OptIdx::kSize):
            if (1 != sscanf(optarg, "%lu", &(opts->size))) {
                fprintf(stderr, "Invalid size: %s\n", optarg);
                parse_err = true;
            } else {
                size_specified = true;
            }
            break;
        case static_cast<int>(OptIdx::kNumWarmUp):
            if (1 != sscanf(optarg, "%lu", &(opts->num_warm_up))) {
                fprintf(stderr, "Invalid num_warm_up: %s\n", optarg);
                parse_err = true;
            } else {
                num_warm_up_specified = true;
            }
            break;
        case static_cast<int>(OptIdx::kNumLoops):
            if (1 != sscanf(optarg, "%lu", &(opts->num_loops))) {
                fprintf(stderr, "Invalid num_loops: %s\n", optarg);
                parse_err = true;
            } else {
                num_loops_specified = true;
            }
            break;
        case static_cast<int>(OptIdx::kAllToAllNumThreadBlocksPerRank):
            if (1 != sscanf(optarg, "%lu", &(opts->all_to_all_num_thread_blocks_per_rank))) {
                fprintf(stderr, "Invalid all_to_all_num_thread_blocks_per_rank: %s\n", optarg);
                parse_err = true;
            }
            break;
        case static_cast<int>(OptIdx::kAllToAllThreadBlockSize):
            if (1 != sscanf(optarg, "%lu", &(opts->all_to_all_thread_block_size))) {
                fprintf(stderr, "Invalid all_to_all_thread_block_size: %s\n", optarg);
                parse_err = true;
            }
            break;
        case static_cast<int>(OptIdx::kEnableSmCopy):
            opts->sm_copy_enabled = true;
            break;
        case static_cast<int>(OptIdx::kEnableDmaCopy):
            opts->dma_copy_enabled = true;
            break;
        case static_cast<int>(OptIdx::kEnableHToD):
            opts->htod_enabled = true;
            break;
        case static_cast<int>(OptIdx::kEnableDToH):
            opts->dtoh_enabled = true;
            break;
        case static_cast<int>(OptIdx::kEnableDToD):
            opts->dtod_enabled = true;
            break;
        case static_cast<int>(OptIdx::kEnableOneToAll):
            opts->one_to_all_enabled = true;
            break;
        case static_cast<int>(OptIdx::kEnableAllToOne):
            opts->all_to_one_enabled = true;
            break;
        case static_cast<int>(OptIdx::kEnableAllToAll):
            opts->all_to_all_enabled = true;
            break;
        case static_cast<int>(OptIdx::kEnableBidirectional):
            opts->bidirectional_enabled = true;
            break;
        case static_cast<int>(OptIdx::kEnableCheckData):
            opts->check_data = true;
            break;
        default:
            parse_err = true;
        }
        if (parse_err) {
            break;
        }
    }
    if (parse_err) {
        PrintUsage();
        return -1;
    }
    return 0;
}

// Get nubmer of GPUs available.
int GetGpuCount(int *gpu_count) {
    hipError_t cuda_err = hipGetDeviceCount(gpu_count);
    if (cuda_err != hipSuccess) {
        fprintf(stderr, "GetGpuCount::hipGetDeviceCount error: %d\n", cuda_err);
        return -1;
    }
    return 0;
}

// Set GPU context according to device ID.
int SetGpu(int gpu_id) {
    hipError_t cuda_err = hipSetDevice(gpu_id);
    if (cuda_err != hipSuccess) {
        fprintf(stderr, "SetGpu::hipSetDevice %d error: %d\n", gpu_id, cuda_err);
        return -1;
    }
    return 0;
}

#if defined(__HIP_PLATFORM_AMD__)
bool UseFineGrained(const SubBenchArgs &args) {
    return args.is_src_dev_gpu && args.is_dst_dev_gpu && args.src_gpu_id != args.dst_gpu_id;
}
cudaError_t GpuMallocDataBuf(uint8_t **ptr, uint64_t size, bool use_fine_grained) {
    if (use_fine_grained) {
#if defined(HIP_UNCACHED_MEMORY)
        return hipExtMallocWithFlags((void **)ptr, size, hipDeviceMallocUncached);
#else
        return hipExtMallocWithFlags((void **)ptr, size, hipDeviceMallocFinegrained);
#endif
    } else {
        return cudaMalloc(ptr, size);
    }
}
#else
hipError_t GpuMallocDataBuf(uint8_t **ptr, uint64_t size) { return hipMalloc(ptr, size); }
#endif

// Prepare data buffers and streams to be used.
int PrepareBufAndStream(BenchArgs *args) {
    hipError_t cuda_err = hipSuccess;
    constexpr int uint8_mod = 256;

    for (int i = 0; i < args->num_subs; i++) {
        SubBenchArgs &sub = args->subs[i];

        if (args->check_data) {
            // Generate data to copy
            sub.data_buf = static_cast<uint8_t *>(numa_alloc_onnode(args->size, args->numa_id));

            for (int j = 0; j < args->size; j++) {
                sub.data_buf[j] = static_cast<uint8_t>(j % uint8_mod);
            }
            // Allocate check buffer
            sub.check_buf = static_cast<uint8_t *>(numa_alloc_onnode(args->size, args->numa_id));
        }

        // Allocate buffers for src/dst devices
        constexpr int num_devices = 2;
        bool is_dev_gpu[num_devices] = {sub.is_src_dev_gpu, sub.is_dst_dev_gpu};
        int dev_ids[num_devices] = {sub.src_gpu_id, sub.dst_gpu_id};
        uint8_t **host_buf_ptrs[num_devices] = {&(sub.src_dev_host_buf_ptr), &(sub.dst_dev_host_buf_ptr)};
        uint8_t **gpu_buf_ptrs[num_devices] = {&(sub.src_dev_gpu_buf_ptr), &(sub.dst_dev_gpu_buf_ptr)};
        for (int j = 0; j < num_devices; j++) {
            // Allocate buffers
            if (is_dev_gpu[j]) {
                // Set to buffer device for GPU buffer
                if (SetGpu(dev_ids[j])) {
                    return -1;
                }
                *(host_buf_ptrs[j]) = nullptr;
#if defined(__HIP_PLATFORM_AMD__)
                cuda_err = GpuMallocDataBuf(gpu_buf_ptrs[j], args->size, UseFineGrained(sub));
#else
                cuda_err = GpuMallocDataBuf(gpu_buf_ptrs[j], args->size);
#endif
                if (cuda_err != hipSuccess) {
                    fprintf(stderr, "PrepareBufAndStream::hipMalloc error: %d\n", cuda_err);
                    return -1;
                }
            } else {
                // Set to worker device for host memory buffer
                if (SetGpu(sub.worker_gpu_id)) {
                    return -1;
                }
                *(host_buf_ptrs[j]) = static_cast<uint8_t *>(numa_alloc_onnode(args->size, args->numa_id));
                cuda_err = hipHostRegister(*(host_buf_ptrs[j]), args->size, hipHostRegisterMapped);
                if (cuda_err != hipSuccess) {
                    fprintf(stderr, "PrepareBufAndStream::hipHostRegister error: %d\n", cuda_err);
                    return -1;
                }
                cuda_err = hipHostGetDevicePointer((void **)gpu_buf_ptrs[j], *(host_buf_ptrs[j]), 0);
                if (cuda_err != hipSuccess) {
                    fprintf(stderr, "PrepareBufAndStream::hipHostGetDevicePointer error: %d\n", cuda_err);
                    return -1;
                }
            }
        }

        // Initialize source buffer
        if (args->check_data) {
            if (sub.is_src_dev_gpu) {
                if (SetGpu(sub.src_gpu_id)) {
                    return -1;
                }
            }
            cuda_err = hipMemcpy(sub.src_dev_gpu_buf_ptr, sub.data_buf, args->size, hipMemcpyDefault);
            if (cuda_err != hipSuccess) {
                fprintf(stderr, "PrepareBufAndStream::hipMemcpy error: %d\n", cuda_err);
                return -1;
            }
        }

        // Initialize stream on worker device
        if (SetGpu(sub.worker_gpu_id)) {
            return -1;
        }
        cuda_err = hipStreamCreateWithFlags(&(sub.stream), hipStreamNonBlocking);
        if (cuda_err != hipSuccess) {
            fprintf(stderr, "PrepareBufAndStream::hipStreamCreate error: %d\n", cuda_err);
            return -1;
        }
    }

    return 0;
}

// Prepare events to be used.
int PrepareEvent(BenchArgs *args) {
    hipError_t cuda_err = hipSuccess;
    for (int i = 0; i < args->num_subs; i++) {
        SubBenchArgs &sub = args->subs[i];
        if (SetGpu(sub.worker_gpu_id)) {
            return -1;
        }
        cuda_err = hipEventCreate(&(sub.start_event));
        if (cuda_err != hipSuccess) {
            fprintf(stderr, "PrepareEvent::hipEventCreate error: %d\n", cuda_err);
            return -1;
        }
        cuda_err = hipEventCreate(&(sub.end_event));
        if (cuda_err != hipSuccess) {
            fprintf(stderr, "PrepareEvent::hipEventCreate error: %d\n", cuda_err);
            return -1;
        }
    }
    return 0;
}

// Validate the result of data transfer.
int CheckBuf(BenchArgs *args) {
    hipError_t cuda_err = hipSuccess;
    int memcmp_result = 0;

    for (int i = 0; i < args->num_subs; i++) {
        SubBenchArgs &sub = args->subs[i];

        // Copy result
        memset(sub.check_buf, 0, args->size);
        if (SetGpu(sub.dst_gpu_id)) {
            return -1;
        }
        cuda_err = hipMemcpy(sub.check_buf, sub.dst_dev_gpu_buf_ptr, args->size, hipMemcpyDefault);
        if (cuda_err != hipSuccess) {
            fprintf(stderr, "CheckBuf::hipMemcpy error: %d\n", cuda_err);
            return -1;
        }

        // Validate result
        memcmp_result = memcmp(sub.data_buf, sub.check_buf, args->size);
        if (memcmp_result) {
            fprintf(stderr, "CheckBuf: Memory check failed\n");
            return -1;
        }
    }

    return 0;
}

// Destroy data buffers and streams
int DestroyBufAndStream(BenchArgs *args) {
    int ret = 0;
    hipError_t cuda_err = hipSuccess;

    for (int i = 0; i < args->num_subs; i++) {
        SubBenchArgs &sub = args->subs[i];

        // Destroy original data buffer and check buffer
        if (sub.data_buf != nullptr) {
            numa_free(sub.data_buf, args->size);
        }
        if (sub.check_buf != nullptr) {
            numa_free(sub.check_buf, args->size);
        }

        // Only destroy buffers for src/dst devices
        constexpr int num_devices = 2;
        bool is_dev_gpu[num_devices] = {sub.is_src_dev_gpu, sub.is_dst_dev_gpu};
        int dev_ids[num_devices] = {sub.src_gpu_id, sub.dst_gpu_id};
        uint8_t **host_buf_ptrs[num_devices] = {&(sub.src_dev_host_buf_ptr), &(sub.dst_dev_host_buf_ptr)};
        uint8_t **gpu_buf_ptrs[num_devices] = {&(sub.src_dev_gpu_buf_ptr), &(sub.dst_dev_gpu_buf_ptr)};
        for (int i = 0; i < num_devices; i++) {
            // Destroy buffers
            if (is_dev_gpu[i]) {
                if (*(gpu_buf_ptrs[i]) == nullptr) {
                    continue;
                }
                // Set to buffer device for GPU buffer
                if (SetGpu(dev_ids[i])) {
                    return -1;
                }
                cuda_err = hipFree(*(gpu_buf_ptrs[i]));
                if (cuda_err != hipSuccess) {
                    fprintf(stderr, "DestroyBufAndStream::hipFree error: %d\n", cuda_err);
                    ret = -1;
                }
                *(gpu_buf_ptrs[i]) = nullptr;
            } else {
                if (*(host_buf_ptrs[i]) == nullptr) {
                    continue;
                }
                // Set to worker device for host memory buffer
                if (SetGpu(sub.worker_gpu_id)) {
                    return -1;
                }
                cuda_err = hipHostUnregister(*(host_buf_ptrs[i]));
                if (cuda_err != hipSuccess) {
                    fprintf(stderr, "DestroyBufAndStream::hipHostUnregister error: %d\n", cuda_err);
                    ret = -1;
                }
                numa_free(*(host_buf_ptrs[i]), args->size);
                *(host_buf_ptrs[i]) = nullptr;
                *(gpu_buf_ptrs[i]) = nullptr;
            }
        }

        // Destroy stream on worker device
        if (SetGpu(sub.worker_gpu_id)) {
            return -1;
        }
        cuda_err = hipStreamDestroy(sub.stream);
        if (cuda_err != hipSuccess) {
            fprintf(stderr, "DestroyBufAndStream::hipStreamDestroy error: %d\n", cuda_err);
            return -1;
        }
    }

    return ret;
}

// Destroy events
int DestroyEvent(BenchArgs *args) {
    hipError_t cuda_err = hipSuccess;
    for (int i = 0; i < args->num_subs; i++) {
        SubBenchArgs &sub = args->subs[i];
        if (SetGpu(sub.worker_gpu_id)) {
            return -1;
        }
        cuda_err = hipEventDestroy(sub.start_event);
        if (cuda_err != hipSuccess) {
            fprintf(stderr, "DestroyEvent::hipEventDestroy error: %d\n", cuda_err);
            return -1;
        }
        cuda_err = hipEventDestroy(sub.end_event);
        if (cuda_err != hipSuccess) {
            fprintf(stderr, "DestroyEvent::hipEventDestroy error: %d\n", cuda_err);
            return -1;
        }
    }
    return 0;
}

// Unroll depth in SM copy kernel
#define NUM_LOOP_UNROLL 2

// Thread block size
#define NUM_THREADS_IN_BLOCK 128

// Fetch a ulong2 from source memory and write to register
// This kernel references the implementation in
// 1) NCCL:
// https://github.com/NVIDIA/nccl/blob/7e515921295adaab72adf56ea71a0fafb0ecb5f3/src/collectives/device/common_kernel.h#L483
// 2) RCCL:
// https://github.com/ROCmSoftwarePlatform/rccl/blob/5c8380ff5b5925cae4bce00b1879a5f930226e8d/src/collectives/device/common_kernel.h#L268
inline __device__ void FetchULong2(ulong2 &v, const ulong2 *p) {
#if defined(__HIP_PLATFORM_HCC__) || defined(__HCC__) || defined(__HIPCC__)
    v.x = p->x;
    v.y = p->y;
#else
    asm volatile("ld.volatile.global.v2.u64 {%0,%1}, [%2];" : "=l"(v.x), "=l"(v.y) : "l"(p) : "memory");
#endif
}

// Store a ulong2 from register and write to target memory
// This kernel references the implementation in
// 1) NCCL:
// https://github.com/NVIDIA/nccl/blob/7e515921295adaab72adf56ea71a0fafb0ecb5f3/src/collectives/device/common_kernel.h#L486
// 2) RCCL:
// https://github.com/ROCmSoftwarePlatform/rccl/blob/5c8380ff5b5925cae4bce00b1879a5f930226e8d/src/collectives/device/common_kernel.h#L276
inline __device__ void StoreULong2(ulong2 *p, ulong2 &v) {
#if defined(__HIP_PLATFORM_HCC__) || defined(__HCC__) || defined(__HIPCC__)
    p->x = v.x;
    p->y = v.y;
#else
    asm volatile("st.volatile.global.v2.u64 [%0], {%1,%2};" ::"l"(p), "l"(v.x), "l"(v.y) : "memory");
#endif
}

// Fetch data from source memory into register first, and then write them to target memory
// Stride set to thread block size to best utilize cache
__global__ void SMCopyKernel(ulong2 *tgt, const ulong2 *src) {
    uint64_t index = blockIdx.x * blockDim.x * NUM_LOOP_UNROLL + threadIdx.x;
    ulong2 val[NUM_LOOP_UNROLL];
#pragma unroll
    for (uint64_t i = 0; i < NUM_LOOP_UNROLL; i++)
        FetchULong2(val[i], src + index + i * blockDim.x);
#pragma unroll
    for (uint64_t i = 0; i < NUM_LOOP_UNROLL; i++)
        StoreULong2(tgt + index + i * blockDim.x, val[i]);
}

// Print result tag as <src_dev>_to_<dst_dev>_by_<worker_dev>_using_<sm|dma>_under_<numa_node>.
void PrintResultTag(const BenchArgs &args) {
    if (args.subs[0].is_src_dev_gpu) {
        printf("gpu%d", args.subs[0].src_gpu_id);
    } else {
        printf("cpu");
    }
    printf("%s", args.num_subs == 1 ? "_to_" : "_and_");
    if (args.subs[0].is_dst_dev_gpu) {
        printf("gpu%d", args.subs[0].dst_gpu_id);
    } else {
        printf("cpu");
    }
    if (args.subs[0].is_src_dev_gpu && args.subs[0].is_dst_dev_gpu &&
        args.subs[0].src_gpu_id != args.subs[0].dst_gpu_id) {
        if (args.subs[0].src_gpu_id == args.subs[0].worker_gpu_id) {
            printf("_write");
        } else {
            printf("_read");
        }
    }
    printf("_by_%s", args.is_sm_copy ? "sm" : "dma");
    if (!args.subs[0].is_src_dev_gpu || !args.subs[0].is_dst_dev_gpu) {
        printf("_under_numa%lu", args.numa_id);
    }
}

// Run copy benchmark.
int RunCopy(BenchArgs *args) {
    hipError_t cuda_err = hipSuccess;
    uint64_t num_thread_blocks;

    // Validate data size for SM copy
    if (args->is_sm_copy) {
        uint64_t num_elements_in_thread_block = NUM_LOOP_UNROLL * NUM_THREADS_IN_BLOCK;
        uint64_t num_bytes_in_thread_block = num_elements_in_thread_block * sizeof(ulong2);
        if (args->size % num_bytes_in_thread_block) {
            fprintf(stderr, "RunCopy: Data size should be multiple of %lu\n", num_bytes_in_thread_block);
            return -1;
        }
        num_thread_blocks = args->size / num_bytes_in_thread_block;
    }

    // Launch jobs and collect running time
    for (int i = 0; i < args->num_loops + args->num_warm_up; i++) {
        for (int j = 0; j < args->num_subs; j++) {
            SubBenchArgs &sub = args->subs[j];
            if (SetGpu(sub.worker_gpu_id)) {
                return -1;
            }
            if (i == args->num_warm_up) {
                cuda_err = hipEventRecord(sub.start_event, sub.stream);
                if (cuda_err != hipSuccess) {
                    fprintf(stderr, "RunCopy::hipEventRecord error: %d\n", cuda_err);
                    return -1;
                }
            }
            if (args->is_sm_copy) {
                SMCopyKernel<<<num_thread_blocks, NUM_THREADS_IN_BLOCK, 0, sub.stream>>>(
                    reinterpret_cast<ulong2 *>(sub.dst_dev_gpu_buf_ptr),
                    reinterpret_cast<ulong2 *>(sub.src_dev_gpu_buf_ptr));
            } else {
                cuda_err = hipMemcpyAsync(sub.dst_dev_gpu_buf_ptr, sub.src_dev_gpu_buf_ptr, args->size,
                                           hipMemcpyDefault, sub.stream);
                if (cuda_err != hipSuccess) {
                    fprintf(stderr, "RunCopy::hipMemcpyAsync error: %d\n", cuda_err);
                    return -1;
                }
            }
            if (i + 1 == args->num_loops + args->num_warm_up) {
                cuda_err = hipEventRecord(sub.end_event, sub.stream);
                if (cuda_err != hipSuccess) {
                    fprintf(stderr, "RunCopy::hipEventRecord error: %d\n", cuda_err);
                    return -1;
                }
            }
        }
    }
    for (int i = 0; i < args->num_subs; i++) {
        SubBenchArgs &sub = args->subs[i];
        cuda_err = hipStreamSynchronize(sub.stream);
        if (cuda_err != hipSuccess) {
            fprintf(stderr, "RunCopy::hipStreamSynchronize error: %d\n", cuda_err);
            return -1;
        }
    }

    // Calculate and display bandwidth if no problem
    float max_time_in_ms = 0;
    for (int i = 0; i < args->num_subs; i++) {
        SubBenchArgs &sub = args->subs[i];
        float time_in_ms = 0;
        cuda_err = hipEventElapsedTime(&time_in_ms, sub.start_event, sub.end_event);
        if (cuda_err != hipSuccess) {
            fprintf(stderr, "RunCopy::hipEventElapsedTime error: %d\n", cuda_err);
            return -1;
        }
        max_time_in_ms = time_in_ms > max_time_in_ms ? time_in_ms : max_time_in_ms;
    }

    PrintResultTag(*args);
    double bw = args->size * args->num_loops * args->num_subs / max_time_in_ms / 1e6;
    if (args->subs[0].is_src_dev_gpu && args->subs[0].is_dst_dev_gpu &&
        args->subs[0].src_gpu_id == args->subs[0].dst_gpu_id) {
        bw *= 2.0;
    }
    printf(" %g\n", bw);

    return 0;
}

// Enable peer access between a GPU pair. Return whether succeeds.
int EnablePeerAccess(int src_gpu_id, int dst_gpu_id, int *can_access) {
    hipError_t cuda_err = hipSuccess;
    if (src_gpu_id == dst_gpu_id) {
        *can_access = 1;
        return 0;
    }
    cuda_err = hipDeviceCanAccessPeer(can_access, src_gpu_id, dst_gpu_id);
    if (cuda_err != hipSuccess) {
        fprintf(stderr, "EnablePeerAccess::hipDeviceCanAccessPeer error: %d\n", cuda_err);
        return -1;
    }
    if (*can_access) {
        if (SetGpu(src_gpu_id)) {
            return -1;
        }
        cuda_err = hipDeviceEnablePeerAccess(dst_gpu_id, 0);
        if (cuda_err != hipErrorPeerAccessAlreadyEnabled && cuda_err != hipSuccess) {
            fprintf(stderr, "EnablePeerAccess::hipDeviceEnablePeerAccess error: %d\n", cuda_err);
            return -1;
        }
    }
    return 0;
}

int RunBench(BenchArgs *args) {
    int ret = 0;
    int destroy_ret = 0;
    ret = PrepareBufAndStream(args);
    if (ret != 0) {
        goto destroy_buf;
    }
    ret = PrepareEvent(args);
    if (ret != 0) {
        goto destroy_event;
    }
    ret = RunCopy(args);
    if (ret == 0 && args->check_data) {
        ret = CheckBuf(args);
    }
destroy_event:
    destroy_ret = DestroyEvent(args);
    if (ret == 0) {
        ret = destroy_ret;
    }
destroy_buf:
    destroy_ret = DestroyBufAndStream(args);
    if (ret == 0) {
        ret = destroy_ret;
    }
    return ret;
}

void SetSubBenchArgsForHToD(int gpu_id, bool is_bidirectional, BenchArgs *args) {
    args->subs[0].is_src_dev_gpu = false;
    args->subs[0].is_dst_dev_gpu = true;
    args->subs[0].dst_gpu_id = gpu_id;
    args->subs[0].worker_gpu_id = gpu_id;
    if (is_bidirectional) {
        args->num_subs = 2;
        args->subs[1].is_src_dev_gpu = true;
        args->subs[1].is_dst_dev_gpu = false;
        args->subs[1].src_gpu_id = gpu_id;
        args->subs[1].worker_gpu_id = gpu_id;
    } else {
        args->num_subs = 1;
    }
}

void SetSubBenchArgsForDToH(int gpu_id, bool is_bidirectional, BenchArgs *args) {
    args->subs[0].is_src_dev_gpu = true;
    args->subs[0].is_dst_dev_gpu = false;
    args->subs[0].src_gpu_id = gpu_id;
    args->subs[0].worker_gpu_id = gpu_id;
    if (is_bidirectional) {
        args->num_subs = 2;
        args->subs[1].is_src_dev_gpu = false;
        args->subs[1].is_dst_dev_gpu = true;
        args->subs[1].dst_gpu_id = gpu_id;
        args->subs[1].worker_gpu_id = gpu_id;
    } else {
        args->num_subs = 1;
    }
}

void SetSubBenchArgsForDToD(int src_gpu_id, int dst_gpu_id, bool is_read, bool is_bidirectional, BenchArgs *args) {
    args->subs[0].is_src_dev_gpu = true;
    args->subs[0].is_dst_dev_gpu = true;
    args->subs[0].src_gpu_id = src_gpu_id;
    args->subs[0].dst_gpu_id = dst_gpu_id;
    args->subs[0].worker_gpu_id = is_read ? dst_gpu_id : src_gpu_id;
    if (is_bidirectional) {
        args->num_subs = 2;
        args->subs[1].is_src_dev_gpu = true;
        args->subs[1].is_dst_dev_gpu = true;
        args->subs[1].src_gpu_id = dst_gpu_id;
        args->subs[1].dst_gpu_id = src_gpu_id;
        args->subs[1].worker_gpu_id = is_read ? src_gpu_id : dst_gpu_id;
    } else {
        args->num_subs = 1;
    }
}

// dst_rank: < 0 for all ranks, else for specified rank
__global__ void SMOneToAllCopyKernel(ulong2 **dst_buffers, ulong2 *src_buffer, uint64_t msg_size, int src_rank,
                                     int dst_rank, int gpu_count) {
    uint64_t size_per_dst_rank = msg_size / gpu_count;
    uint64_t num_blocks_per_dst_rank = gridDim.x / gpu_count;
    if (dst_rank >= 0 && dst_rank != blockIdx.x / num_blocks_per_dst_rank) {
        return;
    } else {
        dst_rank = blockIdx.x / num_blocks_per_dst_rank;
    }
    if (src_rank == dst_rank) {
        return;
    }
    ulong2 *dst_buffer = dst_buffers[dst_rank];
    uint64_t src_rank_elem_offset = uint64_t(src_rank) * size_per_dst_rank / sizeof(ulong2);
    uint64_t dst_rank_elem_offset = uint64_t(dst_rank) * size_per_dst_rank / sizeof(ulong2);
    ulong2 val;

    uint64_t size_per_block = msg_size / gridDim.x;
    uint64_t num_elems_per_block = size_per_block / sizeof(ulong2);
    for (uint64_t idx = num_elems_per_block * blockIdx.x + threadIdx.x; idx < num_elems_per_block * (blockIdx.x + 1);
         idx += blockDim.x) {
        FetchULong2(val, src_buffer + idx);
        StoreULong2(dst_buffer + idx + src_rank_elem_offset - dst_rank_elem_offset, val);
    }
}

// src_rank/dst_rank: < 0 for all ranks, else for specified rank
int RunAllToAllBench(const Opts &opts, int gpu_count, int src_rank, int dst_rank) {
    int ret = 0;
    hipError_t cuda_err = hipSuccess;
    int can_access = 0;

    std::vector<uint8_t *> src_buffers_gpu(gpu_count, nullptr);
    std::vector<uint8_t *> dst_buffers_gpu(gpu_count, nullptr);
    std::vector<uint8_t **> dst_buffer_gpu_args(gpu_count, nullptr);
    std::vector<hipStream_t> streams(gpu_count);
    std::vector<hipEvent_t> start_events(gpu_count);
    std::vector<hipEvent_t> stop_events(gpu_count);

    uint64_t *data_buffer_cpu = nullptr;

    // Scan all GPUs
    for (int i = 0; i < gpu_count; i++) {
        for (int j = 0; j < gpu_count; j++) {
            ret = EnablePeerAccess(i, j, &can_access);
            if (ret != 0) {
                fprintf(stderr, "RunAllToAllBench::EnablePeerAccess between GPU %d and GPU %d failed with %d\n", i, j,
                        ret);
                return -1;
            }
            if (!can_access) {
                fprintf(stderr, "RunAllToAllBench: GPU %d cannot talk with GPU %d\n", i, j);
                return -1;
            }
        }
    }

    // Prepare per-GPU resources
    if (opts.check_data) {
        data_buffer_cpu = new uint64_t[opts.size / sizeof(uint64_t)];
    }

    for (int rank = 0; rank < gpu_count; rank++) {
        if (SetGpu(rank)) {
            fprintf(stderr, "RunAllToAllBench::SetGpu for rank %d error: %d\n", cuda_err, rank);
            return -1;
        }

        // Prepare source buffers
#if defined(__HIP_PLATFORM_AMD__)
        cuda_err = GpuMallocDataBuf(&(src_buffers_gpu[rank]), opts.size, true);
#else
        cuda_err = GpuMallocDataBuf(&(src_buffers_gpu[rank]), opts.size);
#endif
        if (cuda_err != hipSuccess) {
            fprintf(stderr, "RunAllToAllBench::hipMalloc for src_buffers_gpu[%d] error: %d\n", cuda_err, rank);
            return -1;
        }
        if (opts.check_data) {
            for (uint64_t i = 0; i < opts.size / sizeof(uint64_t); i++) {
                data_buffer_cpu[i] = i * rank;
            }
            cuda_err = hipMemcpy(src_buffers_gpu[rank], data_buffer_cpu, opts.size, hipMemcpyDefault);
            if (cuda_err != hipSuccess) {
                fprintf(stderr, "RunAllToAllBench::hipMemcpy to src_buffers_gpu[%d] error: %d\n", cuda_err, rank);
                return -1;
            }
        }

        // Prepare destination buffers
#if defined(__HIP_PLATFORM_AMD__)
        cuda_err = GpuMallocDataBuf(&(dst_buffers_gpu[rank]), opts.size, true);
#else
        cuda_err = GpuMallocDataBuf(&(dst_buffers_gpu[rank]), opts.size);
#endif
        if (cuda_err != hipSuccess) {
            fprintf(stderr, "RunAllToAllBench::hipMalloc for dst_buffers_gpu[%d] error: %d\n", cuda_err, rank);
            return -1;
        }

        // Prepare streams
        cuda_err = hipStreamCreateWithFlags(&(streams[rank]), hipStreamNonBlocking);
        if (cuda_err != hipSuccess) {
            fprintf(stderr, "RunAllToAllBench::hipStreamCreateWithFlags error: %d\n", cuda_err);
            return -1;
        }

        // Prepare events
        cuda_err = hipEventCreate(&(start_events[rank]));
        if (cuda_err != hipSuccess) {
            fprintf(stderr, "RunAllToAllBench::hipEventCreate for start_events[%d] error: %d\n", cuda_err, rank);
            return -1;
        }
        cuda_err = hipEventCreate(&(stop_events[rank]));
        if (cuda_err != hipSuccess) {
            fprintf(stderr, "RunAllToAllBench::hipEventCreate for stop_events[%d] error: %d\n", cuda_err, rank);
            return -1;
        }
    }

    // Prepare kernel arguments
    for (int rank = 0; rank < gpu_count; rank++) {
        if (SetGpu(rank)) {
            fprintf(stderr, "RunAllToAllBench::SetGpu for rank %d error: %d\n", cuda_err, rank);
            return -1;
        }

        // Prepare destination buffer args
        cuda_err = hipMalloc(&(dst_buffer_gpu_args[rank]), sizeof(uint8_t *) * gpu_count);
        if (cuda_err != hipSuccess) {
            fprintf(stderr, "RunAllToAllBench::hipMalloc for dst_buffer_gpu_args[%d] error: %d\n", cuda_err, rank);
            return -1;
        }
        cuda_err = hipMemcpy(dst_buffer_gpu_args[rank], dst_buffers_gpu.data(), sizeof(uint8_t *) * gpu_count,
                              hipMemcpyDefault);
        if (cuda_err != hipSuccess) {
            fprintf(stderr, "RunAllToAllBench::hipMemcpy to dst_buffer_gpu_args[%d] error: %d\n", cuda_err, rank);
            return -1;
        }
    }

    // Launch jobs and collect running time
    for (int i = 0; i < opts.num_warm_up + opts.num_loops; i++) {
        for (int rank = 0; rank < gpu_count; rank++) {
            if (src_rank >= 0 && rank != src_rank) {
                continue;
            }

            if (SetGpu(rank)) {
                fprintf(stderr, "RunAllToAllBench::SetGpu for rank %d error: %d\n", cuda_err, rank);
                return -1;
            }

            if (i == opts.num_warm_up) {
                cuda_err = hipEventRecord(start_events[rank], streams[rank]);
                if (cuda_err != hipSuccess) {
                    fprintf(stderr, "RunAllToAllBench::hipEventRecord for start_events[%d] error: %d\n", cuda_err,
                            rank);
                    return -1;
                }
            }
            SMOneToAllCopyKernel<<<gpu_count * opts.all_to_all_num_thread_blocks_per_rank,
                                   opts.all_to_all_thread_block_size, 0, streams[rank]>>>(
                (ulong2 **)dst_buffer_gpu_args[rank], (ulong2 *)src_buffers_gpu[rank], opts.size, rank, dst_rank,
                gpu_count);
            if (i == opts.num_warm_up + opts.num_loops - 1) {
                cuda_err = hipEventRecord(stop_events[rank], streams[rank]);
                if (cuda_err != hipSuccess) {
                    fprintf(stderr, "RunAllToAllBench::hipEventRecord for stop_events[%d] error: %d\n", cuda_err,
                            rank);
                    return -1;
                }
            }
        }
    }
    for (int rank = 0; rank < gpu_count; rank++) {
        if (src_rank >= 0 && rank != src_rank) {
            continue;
        }
        cuda_err = hipStreamSynchronize(streams[rank]);
        if (cuda_err != hipSuccess) {
            fprintf(stderr, "RunAllToAllBench::hipStreamSynchronize streams[%d] error: %d\n", cuda_err, rank);
            return -1;
        }
    }

    // Calculate and display bandwidth if no problem
    bool first_bw_seen = false;
    double min_bw = 0.;
    for (int rank = 0; rank < gpu_count; rank++) {
        if (src_rank >= 0 && rank != src_rank) {
            continue;
        }
        float time_in_ms = 0;
        cuda_err = hipEventElapsedTime(&time_in_ms, start_events[rank], stop_events[rank]);
        if (cuda_err != hipSuccess) {
            fprintf(stderr, "RunAllToAllBench::hipEventElapsedTime for rank %d error: %d\n", cuda_err, rank);
            return -1;
        }
        double bw = opts.size * (gpu_count - 1) * opts.num_loops / gpu_count / time_in_ms / 1e6;
        if (!first_bw_seen) {
            min_bw = bw;
            first_bw_seen = true;
        } else {
            min_bw = std::min(min_bw, bw);
        }
    }
    if (src_rank < 0 && dst_rank < 0) {
        printf("gpu_all_to_gpu_all_write_by_sm %g\n", min_bw);
    } else if (src_rank < 0) {
        printf("gpu_all_to_gpu%d_write_by_sm %g\n", dst_rank, min_bw);
    } else {
        printf("gpu%d_to_gpu_all_write_by_sm %g\n", src_rank, min_bw);
    }

    // Check data
    if (opts.check_data) {
        for (int curr_dst_rank = 0; curr_dst_rank < gpu_count; curr_dst_rank++) {
            if (dst_rank >= 0 && dst_rank != curr_dst_rank) {
                continue;
            }
            cuda_err = hipMemcpy(data_buffer_cpu, dst_buffers_gpu[curr_dst_rank], opts.size, hipMemcpyDefault);
            if (cuda_err != hipSuccess) {
                fprintf(stderr, "RunAllToAllBench::hipMemcpy from dst_buffers_gpu[%d] error: %d\n", cuda_err,
                        curr_dst_rank);
                return -1;
            }
            for (uint64_t i = 0; i < opts.size / sizeof(uint64_t); i++) {
                int curr_src_rank = i / (opts.size / sizeof(uint64_t) / gpu_count);
                if (src_rank >= 0 && src_rank != curr_src_rank) {
                    continue;
                }
                if (curr_src_rank == curr_dst_rank) {
                    continue;
                }
                uint64_t offset_in_src_rank = (i % (opts.size / sizeof(uint64_t) / gpu_count)) +
                                              curr_dst_rank * (opts.size / sizeof(uint64_t) / gpu_count);
                if (data_buffer_cpu[i] != offset_in_src_rank * curr_src_rank) {
                    fprintf(stderr,
                            "RunAllToAllBench: data check failure, dst_buffers_gpu[%d][%lu] (%lu) != %lu * %d\n",
                            curr_dst_rank, i, data_buffer_cpu[i], offset_in_src_rank, curr_src_rank);
                    return -1;
                }
            }
        }
    }

    // Free buffers
    for (int rank = 0; rank < gpu_count; rank++) {
        cuda_err = hipFree(src_buffers_gpu[rank]);
        if (cuda_err != hipSuccess) {
            fprintf(stderr, "RunAllToAllBench::hipFree for src_buffers_gpu[%d] error: %d\n", cuda_err, rank);
            return -1;
        }
        cuda_err = hipFree(dst_buffers_gpu[rank]);
        if (cuda_err != hipSuccess) {
            fprintf(stderr, "RunAllToAllBench::hipFree for dst_buffers_gpu[%d] error: %d\n", cuda_err, rank);
            return -1;
        }
        cuda_err = hipFree(dst_buffer_gpu_args[rank]);
        if (cuda_err != hipSuccess) {
            fprintf(stderr, "RunAllToAllBench::hipFree for dst_buffer_gpu_args[%d] error: %d\n", cuda_err, rank);
            return -1;
        }
    }
    delete[] data_buffer_cpu;

    return 0;
}

int main(int argc, char **argv) {
    int ret = 0;
    int numa_count = 0;
    int gpu_count = 0;
    Opts opts;
    BenchArgs args;
    std::vector<BenchArgs> args_list;
    int can_access = 0;

    ret = ParseOpts(argc, argv, &opts);
    if (ret != 0) {
        return ret;
    }
    args.num_warm_up = opts.num_warm_up;
    args.num_loops = opts.num_loops;
    args.size = opts.size;
    args.check_data = opts.check_data;

    // Get number of NUMA nodes
    if (numa_available()) {
        fprintf(stderr, "main::numa_available error\n");
        return -1;
    }
    numa_count = numa_num_configured_nodes();

    // Get number of GPUs
    ret = GetGpuCount(&gpu_count);
    if (ret != 0) {
        return ret;
    }

    // Scan all NUMA nodes
    for (int i = 0; i < numa_count; i++) {
        args.numa_id = i;
        // Scan all GPUs
        for (int j = 0; j < gpu_count; j++) {
            // Host-to-device benchmark
            if (opts.htod_enabled) {
                if (opts.sm_copy_enabled) {
                    args.is_sm_copy = true;
                    SetSubBenchArgsForHToD(j, opts.bidirectional_enabled, &args);
                    args_list.push_back(args);
                }
                if (opts.dma_copy_enabled) {
                    args.is_sm_copy = false;
                    SetSubBenchArgsForHToD(j, opts.bidirectional_enabled, &args);
                    args_list.push_back(args);
                }
            }
            // Device-to-host benchmark
            if (opts.dtoh_enabled) {
                if (opts.sm_copy_enabled) {
                    args.is_sm_copy = true;
                    SetSubBenchArgsForDToH(j, opts.bidirectional_enabled, &args);
                    args_list.push_back(args);
                }
                if (opts.dma_copy_enabled) {
                    args.is_sm_copy = false;
                    SetSubBenchArgsForDToH(j, opts.bidirectional_enabled, &args);
                    args_list.push_back(args);
                }
            }
            if (args.numa_id != 0) {
                continue;
            }
            // Device-to-device benchmark
            if (opts.dtod_enabled) {
                // Scan all peers
                for (int k = 0; k < gpu_count; k++) {
                    // src_dev_id always <= dst_dev_id for bidirectional test
                    if (opts.bidirectional_enabled && j > k) {
                        continue;
                    }
                    // P2P write
                    ret = EnablePeerAccess(j, k, &can_access);
                    if (ret != 0) {
                        return -1;
                    }
                    if (can_access) {
                        if (opts.sm_copy_enabled) {
                            args.is_sm_copy = true;
                            SetSubBenchArgsForDToD(j, k, false, opts.bidirectional_enabled, &args);
                            args_list.push_back(args);
                        }
                        if (opts.dma_copy_enabled) {
                            args.is_sm_copy = false;
                            SetSubBenchArgsForDToD(j, k, false, opts.bidirectional_enabled, &args);
                            args_list.push_back(args);
                        }
                    }
                    if (j == k) {
                        continue;
                    }
                    // P2P read
                    ret = EnablePeerAccess(k, j, &can_access);
                    if (ret != 0) {
                        return -1;
                    }
                    if (can_access) {
                        if (opts.sm_copy_enabled) {
                            args.is_sm_copy = true;
                            SetSubBenchArgsForDToD(j, k, true, opts.bidirectional_enabled, &args);
                            args_list.push_back(args);
                        }
                        if (opts.dma_copy_enabled) {
                            args.is_sm_copy = false;
                            SetSubBenchArgsForDToD(j, k, true, opts.bidirectional_enabled, &args);
                            args_list.push_back(args);
                        }
                    }
                }
            }
        }
    }

    for (BenchArgs &curr_args : args_list) {
        ret = numa_run_on_node(curr_args.numa_id);
        if (ret != 0) {
            fprintf(stderr, "main::numa_run_on_node error: %d\n", errno);
            return -1;
        }
        ret = RunBench(&curr_args);
        if (ret != 0) {
            return -1;
        }
    }

    if (opts.one_to_all_enabled) {
        for (int i = 0; i < gpu_count; i++) {
            ret = RunAllToAllBench(opts, gpu_count, i, -1);
            if (ret != 0) {
                return -1;
            }
        }
    }

    if (opts.all_to_one_enabled) {
        for (int i = 0; i < gpu_count; i++) {
            ret = RunAllToAllBench(opts, gpu_count, -1, i);
            if (ret != 0) {
                return -1;
            }
        }
    }

    if (opts.all_to_all_enabled) {
        ret = RunAllToAllBench(opts, gpu_count, -1, -1);
        if (ret != 0) {
            return -1;
        }
    }

    return ret;
}
