// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

#include "cublas_benchmark.h"

// Cuda context init
void cuda_init() {
    CUDA_SAFE_CALL(hipDeviceReset());
    CUDA_SAFE_CALL(hipSetDevice(0));
    // create streams/handles
    CUBLAS_SAFE_CALL(hipblasCreate(&cublas_handle));
}

// Cuda context free
void cuda_free() {
    CUDA_SAFE_CALL(hipSetDevice(0));
    CUBLAS_SAFE_CALL(hipblasDestroy(cublas_handle));
}

template <>
void gemm<float>(hipblasHandle_t handle, int transa, int transb, int m, int n, int k, const float *a, const float *b,
                 float *c) {
    float alpha = 1.0f;
    float beta = 1.0f;
    CUBLAS_SAFE_CALL(hipblasSgemm(handle, (transa ? HIPBLAS_OP_T : HIPBLAS_OP_N), (transb ? HIPBLAS_OP_T : HIPBLAS_OP_N), m,
                                 n, k, &alpha, a, (transa ? k : m), b, (transb ? n : k), &beta, c, m));
}

template <>
void gemm<double>(hipblasHandle_t handle, int transa, int transb, int m, int n, int k, const double *a, const double *b,
                  double *c) {
    double alpha = 1.0;
    double beta = 1.0;
    CUBLAS_SAFE_CALL(hipblasDgemm(handle, (transa ? HIPBLAS_OP_T : HIPBLAS_OP_N), (transb ? HIPBLAS_OP_T : HIPBLAS_OP_N), m,
                                 n, k, &alpha, a, (transa ? k : m), b, (transb ? n : k), &beta, c, m));
}

template <>
void gemm<hipComplex>(hipblasHandle_t handle, int transa, int transb, int m, int n, int k, const hipComplex *a,
                     const hipComplex *b, hipComplex *c) {
    hipComplex alpha = make_hipComplex(1.0f, 0.0f);
    hipComplex beta = make_hipComplex(0.0f, 0.0f);
    CUBLAS_SAFE_CALL(hipblasCgemm(handle, (transa ? HIPBLAS_OP_T : HIPBLAS_OP_N), (transb ? HIPBLAS_OP_T : HIPBLAS_OP_N), m,
                                 n, k, &alpha, a, (transa ? k : m), b, (transb ? n : k), &beta, c, m));
}

template <>
void gemm<hipDoubleComplex>(hipblasHandle_t handle, int transa, int transb, int m, int n, int k, const hipDoubleComplex *a,
                           const hipDoubleComplex *b, hipDoubleComplex *c) {
    hipDoubleComplex alpha = make_hipDoubleComplex(1.0, 0.0);
    hipDoubleComplex beta = make_hipDoubleComplex(1.0, 0.0);
    CUBLAS_SAFE_CALL(hipblasZgemm(handle, (transa ? HIPBLAS_OP_T : HIPBLAS_OP_N), (transb ? HIPBLAS_OP_T : HIPBLAS_OP_N), m,
                                 n, k, &alpha, a, (transa ? k : m), b, (transb ? n : k), &beta, c, m));
}

void gemmEx(hipblasHandle_t handle, int transa, int transb, int m, int n, int k, const void *a, const void *b, void *c,
            std::string type, bool use_tensor_core) {
    float alpha = 1.0f;
    float beta = 0.0f;
    hipblasComputeType_t compute_type = HIPBLAS_COMPUTE_32F;
    hipDataType matrix_type;
    hipblasGemmAlgo_t algo;
    algo = (use_tensor_core ? CUBLAS_GEMM_DFALT_TENSOR_OP : HIPBLAS_GEMM_DEFAULT);
    if (type.compare("float")) {
        matrix_type = HIP_R_32F;
        CUBLAS_SAFE_CALL(hipblasGemmEx(handle, (transa ? HIPBLAS_OP_T : HIPBLAS_OP_N),
                                      (transb ? HIPBLAS_OP_T : HIPBLAS_OP_N), m, n, k, &alpha, a, matrix_type,
                                      (transa ? k : m), b, matrix_type, (transb ? n : k), &beta, c, matrix_type, m,
                                      compute_type, algo));
    }
    if (type.compare("half")) {
        matrix_type = HIP_R_16F;
        CUBLAS_SAFE_CALL(hipblasGemmEx(handle, (transa ? HIPBLAS_OP_T : HIPBLAS_OP_N),
                                      (transb ? HIPBLAS_OP_T : HIPBLAS_OP_N), m, n, k, &alpha, a, matrix_type,
                                      (transa ? k : m), b, matrix_type, (transb ? n : k), &beta, c, matrix_type, m,
                                      compute_type, algo));
    }
}

void gemmStridedBatchedEx(hipblasHandle_t handle, int transa, int transb, int m, int n, int k, const void *a,
                          const void *b, void *c, std::string type, bool use_tensor_core, int batchCount) {
    float alpha = 1.0f;
    float beta = 1.0f;
    hipblasComputeType_t compute_type = HIPBLAS_COMPUTE_32F;
    hipDataType matrix_type;
    hipblasGemmAlgo_t algo;
    algo = (use_tensor_core ? CUBLAS_GEMM_DFALT_TENSOR_OP : HIPBLAS_GEMM_DEFAULT);
    if (type.compare("float")) {
        matrix_type = HIP_R_32F;
        CUBLAS_SAFE_CALL(hipblasGemmStridedBatchedEx(
            handle, (transa ? HIPBLAS_OP_T : HIPBLAS_OP_N), (transb ? HIPBLAS_OP_T : HIPBLAS_OP_N), m, n, k, &alpha, a,
            matrix_type, (transa ? k : m), m * k, b, matrix_type, (transb ? n : k), n * k, &beta, c, matrix_type, m,
            m * n, batchCount, compute_type, algo));
    }
    if (type.compare("half")) {
        matrix_type = HIP_R_16F;
        CUBLAS_SAFE_CALL(hipblasGemmStridedBatchedEx(
            handle, (transa ? HIPBLAS_OP_T : HIPBLAS_OP_N), (transb ? HIPBLAS_OP_T : HIPBLAS_OP_N), m, n, k, &alpha, a,
            matrix_type, (transa ? k : m), m * k, b, matrix_type, (transb ? n : k), n * k, &beta, c, matrix_type, m,
            m * n, batchCount, compute_type, algo));
    }
}

void sgemmStridedBatched(hipblasHandle_t handle, int transa, int transb, int m, int n, int k, const float *a,
                         const float *b, float *c, int batchCount) {
    float alpha = 1.0f;
    float beta = 1.0f;
    CUBLAS_SAFE_CALL(hipblasSgemmStridedBatched(
        handle, (transa ? HIPBLAS_OP_T : HIPBLAS_OP_N), (transb ? HIPBLAS_OP_T : HIPBLAS_OP_N), m, n, k, &alpha, a,
        (transa ? k : m), m * k, b, (transb ? n : k), n * k, &beta, c, m, m * n, batchCount));
}

void cgemm3mStridedBatched(hipblasHandle_t handle, int transa, int transb, int m, int n, int k, const hipComplex *a,
                           const hipComplex *b, hipComplex *c, int batchCount) {
    hipComplex alpha = make_hipComplex(1.0f, 0.0f);
    hipComplex beta = make_hipComplex(0.0f, 0.0f);
    CUBLAS_SAFE_CALL(cublasCgemm3mStridedBatched(
        handle, (transa ? HIPBLAS_OP_T : HIPBLAS_OP_N), (transb ? HIPBLAS_OP_T : HIPBLAS_OP_N), m, n, k, &alpha, a,
        (transa ? k : m), m * k, b, (transb ? n : k), n * k, &beta, c, m, m * n, batchCount));
}
